
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add two numbers
__global__ void addNumbers(int *a, int *b, int *result) {
    *result = *a + *b;
}

int main() {
    int host_a = 5, host_b = 10, host_result; // Host variables
    int *dev_a, *dev_b, *dev_result;         // Device pointers

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, sizeof(int));
    hipMalloc((void**)&dev_b, sizeof(int));
    hipMalloc((void**)&dev_result, sizeof(int));

    // Copy input data from host to device
    hipMemcpy(dev_a, &host_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &host_b, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with 1 block and 1 thread
    addNumbers<<<1, 1>>>(dev_a, dev_b, dev_result);

    // Copy the result from device to host
    hipMemcpy(&host_result, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    printf("The sum of %d and %d is %d\n", host_a, host_b, host_result);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    return 0;
}
